#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#include "nvtx3/nvToolsExt.h"

#include "./01_naive_gemm.cu"
#include "./02_memory_coalesced_gemm.cu"
#include "./03_shared_memory_gemm.cu"
#include "./04_1d_block_tiling.cu"
#include "./05_2d_block_tiling.cu"
#include "./06_vectorize_gemm.cu"

#define CEIL_DIV(x,y) (((x) + (y) - 1) / (y)) 
#define BLOCK_SIZE 32

#define CUDA_CHECK_ERROR(call) do { \
   hipError_t err = call; \
   if (err != hipSuccess) { \
       printf("CUDA error %s at line %d: %s\n", #call, __LINE__, hipGetErrorString(err)); \
       exit(1); \
   } \
} while(0)

void randomInit(float *data, int size) {
    for (int i = 0; i < size; ++i)
        //data[i] = rand() / (float)RAND_MAX;
        data[i] = 1.0;
}

// Verification
bool verifyResults(float* gpu_cu, float* gpu, int size, float tolerance=1e-5) {
   for(int i = 0; i < size; i++) {
       if(fabs(gpu_cu[i] - gpu[i]) > tolerance) {
           printf("Mismatch at %d: cuBLAS=%f, Manual implementation=%f\n", i, gpu_cu[i], gpu[i]);
           return false;
       }
   }
   return true;
}

void runCublas(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C_cublas) {
    // Run cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                n, m, k,
                &alpha,
                d_B, n,
                d_A, k, 
                &beta,
                d_C_cublas, n);
    hipblasDestroy(handle);
}

void runGemmNaive(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(CEIL_DIV(m, 32), CEIL_DIV(n, 32));
    gemmNaive<<<gridSize, blockSize>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
}

void runGemmMemCoalesced(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
    dim3 blockSize(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridSize(CEIL_DIV(m, 32), CEIL_DIV(n, 32));
    gemmMemCoalesced<BLOCK_SIZE><<<gridSize, blockSize>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
}

void  runGemmSharedMem(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
    dim3 blockSize(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridSize(CEIL_DIV(m, 32), CEIL_DIV(n, 32));
    gemmSharedMem<BLOCK_SIZE><<<gridSize, blockSize>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
}

void  runGemm1dBlockTiling(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
    const uint BM = 64;
    const uint BN = 64;
    const uint BK = 8;
    const uint TM = 8; 
    dim3 blockSize((BM * BN) / TM);
    dim3 gridSize(CEIL_DIV(m, BM), CEIL_DIV(n, BN));
    gemm1dBlockTiling<BM, BN, BK, TM><<<gridSize, blockSize>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
}

void  runGemm2dBlockTiling(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
    const uint BM = 128;
    const uint BN = 128;
    const uint BK = 8;
    const uint TM = 8;
    const uint TN = 8;
    dim3 blockSize((BM * BN) / (TM * TN));
    dim3 gridSize(CEIL_DIV(m, BN), CEIL_DIV(n, BM));
    gemm2dBlockTiling<BM, BN, BK, TM, TN><<<gridSize, blockSize>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
}

void  runGemmVec2dBlockTiling(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
    const uint BM = 128;
    const uint BN = 128;
    const uint BK = 8;
    const uint TM = 8;
    const uint TN = 8;
    dim3 blockSize((BM * BN) / (TM * TN));
    dim3 gridSize(CEIL_DIV(m, BN), CEIL_DIV(n, BM));
    gemmVec2dBlockTiling<BM, BN, BK, TM, TN><<<gridSize, blockSize>>>(m, n, k, alpha, d_A, d_B, beta, d_C);
}

int main(int argc, char **argv) {

    hipDeviceReset();

    nvtxRangePush("Matrix Multiplication");
     
    const float alpha = 1.0f;
    const float beta = 0.0f;

    if (argc != 5) {
        printf("Usage: %s <choice> <m> <n> <k>\n", argv[0]);
        return -1;
    }

    int x = atoi(argv[1]);
    int m = atoi(argv[2]);
    int n = atoi(argv[3]);
    int k = atoi(argv[4]);

    // Allocate host memory
    float *h_A = (float*)malloc(m * k * sizeof(float));
    float *h_B = (float*)malloc(k * n * sizeof(float));
    float *h_C = (float*)malloc(m * n * sizeof(float));
    float *h_C_cublas = (float*)malloc(m * n * sizeof(float));

    // Initialize host matrices
    srand(time(NULL));
    randomInit(h_A, m * n);
    randomInit(h_B, n * k);

    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_C_cublas;
    nvtxRangePush("Memory Allocation");
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_A, m * k * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_B, k * n * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_C, m * n * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_C_cublas, m * n * sizeof(float)));
    nvtxRangePop();

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));

    nvtxRangePush("Memory copy H2D");
    // Copy matrices to device
    CUDA_CHECK_ERROR(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));
    nvtxRangePop();

    runCublas(m, n, k, alpha, d_A, d_B, beta, d_C_cublas);
    
    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    nvtxRangePush("Memory copy D2H - cuBLAS");
    CUDA_CHECK_ERROR(hipMemcpy(h_C_cublas, d_C_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
    nvtxRangePop();

    switch(x) {
        case 1:
            printf("Naive GEMM Kernel:\n");
            CUDA_CHECK_ERROR(hipEventRecord(start));
            runGemmNaive(m, n, k, alpha, d_A, d_B, beta, d_C);
            CUDA_CHECK_ERROR(hipEventRecord(stop));
            CUDA_CHECK_ERROR(hipEventSynchronize(stop));
            break;
        case 2:
            printf("Global Memory Coalescing:\n");
            CUDA_CHECK_ERROR(hipEventRecord(start));
            runGemmMemCoalesced(m, n, k, alpha, d_A, d_B, beta, d_C);
            CUDA_CHECK_ERROR(hipEventRecord(stop));
            CUDA_CHECK_ERROR(hipEventSynchronize(stop));
            break;
        case 3:
            printf("Shared Memory Cache-Blocking:\n");
            CUDA_CHECK_ERROR(hipEventRecord(start));
            runGemmSharedMem(m, n, k, alpha, d_A, d_B, beta, d_C);
            CUDA_CHECK_ERROR(hipEventRecord(stop));
            CUDA_CHECK_ERROR(hipEventSynchronize(stop));
            break;
        case 4:
            printf("1D Block tiling:\n");
            CUDA_CHECK_ERROR(hipEventRecord(start));
            runGemm1dBlockTiling(m, n, k, alpha, d_A, d_B, beta, d_C);
            CUDA_CHECK_ERROR(hipEventRecord(stop));
            CUDA_CHECK_ERROR(hipEventSynchronize(stop));
            break;
        case 5:
            printf("2D Block tiling:\n");
            CUDA_CHECK_ERROR(hipEventRecord(start));
            runGemm2dBlockTiling(m, n, k, alpha, d_A, d_B, beta, d_C);
            CUDA_CHECK_ERROR(hipEventRecord(stop));
            CUDA_CHECK_ERROR(hipEventSynchronize(stop));
            break;
        case 6:
            printf("Vector - 2D Block tiling:\n");
            CUDA_CHECK_ERROR(hipEventRecord(start));
            runGemmVec2dBlockTiling(m, n, k, alpha, d_A, d_B, beta, d_C);
            CUDA_CHECK_ERROR(hipEventRecord(stop));
            CUDA_CHECK_ERROR(hipEventSynchronize(stop));
            break;
        default:
            runGemmNaive(m, n, k, alpha, d_A, d_B, beta, d_C);
            break;
    }

    // Copy result back to host
    nvtxRangePush("Memory copy D2H - Implemented Kernel");
    CUDA_CHECK_ERROR(hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost));
    nvtxRangePop();

    float cuda_time;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&cuda_time, start, stop));

    printf("CUDA kernel time: %.4f ms\n", cuda_time);

    bool match = verifyResults(h_C_cublas, h_C, m * n);

    printf("Results match : %s \n",((match)? ("Yes"):("No")));

    // Add this debug printing
    // printf("Initial values after malloc:\n");
    // printf("h_C first 10 elements: ");
    // for(int i = 0; i < 10 && i < m * n; i++) {
    //     printf("%.2f ", h_C[i]);
    // }
    // printf("\n");

    // printf("h_C_cublas first 10 elements: ");
    // for(int i = 0; i < 10 && i < m * n; i++) {
    //     printf("%.2f ", h_C_cublas[i]);
    // }
    // printf("\n\n");

    // Cleanup
    nvtxRangePush("Free memory");
    CUDA_CHECK_ERROR(hipFree(d_A));
    CUDA_CHECK_ERROR(hipFree(d_B));
    CUDA_CHECK_ERROR(hipFree(d_C));
    CUDA_CHECK_ERROR(hipFree(d_C_cublas));
    nvtxRangePop();
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cublas);

    nvtxRangePush("Destroy events");
    CUDA_CHECK_ERROR(hipEventDestroy(start));
    CUDA_CHECK_ERROR(hipEventDestroy(stop));
    nvtxRangePop();

    nvtxRangePop();


    return 0;
}