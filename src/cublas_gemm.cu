#include <hip/hip_runtime.h>
#include <hipblas.h>

void runCublas(int m, int n, int k, float alpha, float* d_A, float* d_B, float beta, float* d_C) {
   hipblasHandle_t handle;
   hipblasCreate(&handle);
   
   hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               n, m, k,  // Corrected order: n,m,k for cuBLAS
               &alpha,
               d_B, n,   // Leading dimension is n
               d_A, k,   // Leading dimension is k
               &beta,
               d_C, n);  // Leading dimension is n
               
   hipblasDestroy(handle);
}